#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <time.h>
#include "headers.h"

// GPU macro
#define THREADS_PER_BLOCK 32
typedef float floatT;

__global__ void gpu_naive(const int size, const int x, const floatT t, const floatT sigma,
	const int idx, const int idy, const int k_beg, const int k_end, 
	floatT *e, floatT *hx, floatT *hy) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	
	for (int k=k_beg; k<=k_end; k++) {
		if (i>0 && i<(size-1) && j>0 && j<(size-1)) {
			e[INDX(i,j,size)] += (hy[INDX(i,j,size-1)]-hy[INDX(i-1,j,size-1)])- (hx[INDX(i,j,size)]-hx[INDX(i,j-1,size)]);
			if (i==idx && j==idy) {
				e[INDX(i,j,size)] -= FJ(k, x, t, sigma);
			}
		}
		__threadfence();

		if (i<(size-1) && j<size) {
			hy[INDX(i,j,size-1)] += 0.5*(e[INDX(i+1,j,size)]-e[INDX(i,j,size)]);
		}
		if (i<size && j<size-1) {
			hx[INDX(i, j, size)] -= 0.5*(e[INDX(i, j+1, size)] - e[INDX(i, j, size)]);
		}
		__threadfence();
	}
		
}

void host_fdtd(const int size, const int x, const floatT t, const floatT sigma,
    const int idx, const int idy, const int k_beg, const int k_end, 
    floatT *e, floatT *hx, floatT *hy) {
	for (int k = k_beg; k <= k_end; k++) {
		for (int i = 1; i < (size-1); i++) {
			for (int j = 1; j < (size-1); j++) {
				e[INDX(i, j, size)] += (hy[INDX(i, j, (size-1))] - hy[INDX(i-1, j, (size-1))])
				- (hx[INDX(i, j, size)] - hx[INDX(i, j-1, size)]);
			}
		}
		e[INDX(idx, idy, size)] -= FJ(k, x, t, sigma);


		for (int i = 0; i < (size-1); i++) {
			for (int j = 0; j < size; j++) {
				hy[INDX(i,j,(size-1))] += 0.5 * (e[INDX(i+1, j, size)] - e[INDX(i, j, size)]);
			}
		}

		for (int i = 0; i < size; i++) {
			for (int j = 0; j < (size-1); j++) {
				hx[INDX(i, j, size)] -= 0.5 * (e[INDX(i, j+1, size)] - e[INDX(i, j, size)]);
			}
		}
	}
}

int main(int argc, char *argv[]) {

	int dev;
	hipDeviceProp_t deviceProp;
	checkCUDA( hipGetDevice( &dev ) );
	checkCUDA( hipGetDeviceProperties( &deviceProp, dev ) );
	printf("Using GPU %d: %s\n", dev, deviceProp.name );
	
	floatT L = 80.0;
	floatT hx = 1.0;
	floatT ht = hx/sqrt(2.0)/3;
 	floatT sigma = 200*ht;

	fprintf(stdout, "fj output is %f\n", FJ(500, hx, ht, sigma));

	int size = int(2*L/hx)+1;
	int idx = int(1.25*L/hx)+1;
	int idy = int(L/hx)+1;
	fprintf(stdout, "size if %d, source is at idx=%d and idy=%d.\n", size, idx, idy);

	floatT *h_E, *h_Hx, *h_Hy;

	size_t num_E = size * size;
	size_t num_H = (size - 1)*size;	
	size_t numbytes_E = num_E*sizeof(floatT);
	size_t numbytes_H = num_H*sizeof(floatT);
		
	fprintf(stdout, "total memory allocated is %lu\n", numbytes_E+2*numbytes_H);
	
	clock_t t_begin, t_end;	
	t_begin = clock();
	h_E = (floatT *) calloc (num_E, sizeof(floatT));
	h_Hx = (floatT *) calloc (num_H, sizeof(floatT));
	h_Hy = (floatT *) calloc (num_H, sizeof(floatT));

	h_E[INDX(idx, idy, size)] = - FJ(1, hx, ht, sigma);
	
	// GPU memory allocation and initialization
	floatT *d_E, *d_Hx, *d_Hy;
	checkCUDA( hipMalloc( (void **) &d_E, numbytes_E ) );
	checkCUDA( hipMalloc( (void **) &d_Hx, numbytes_H ) );
	checkCUDA( hipMalloc( (void **) &d_Hy, numbytes_H ) );

	checkCUDA( hipMemcpy(d_E, h_E, numbytes_E, hipMemcpyHostToDevice) );
	checkCUDA( hipMemset(d_Hx, 0, numbytes_H) );
	checkCUDA( hipMemset(d_Hy, 0, numbytes_H) );
	
	t_end = clock();
	fprintf(stdout, "Memory allocation time is %f s\n", (float)(t_end - t_begin) / CLOCKS_PER_SEC);

	int k_beg = 2;
	int k_end = 1500;
	
	t_begin = clock();
	host_fdtd(size, hx, ht, sigma, idx, idy, k_beg, k_end, h_E, h_Hx, h_Hy);
	t_end = clock();
	fprintf(stdout, "CPU calculation time for %d iteration is %f s\n", k_end, (float)(t_end - t_begin) / CLOCKS_PER_SEC);
	
	// GPU execution
	
	dim3 threads( THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
	dim3 blocks( (size/threads.x)+1, (size/threads.y)+1, 1);
	fprintf(stdout, "block size is %d by %d.\n", blocks.x, blocks.y);

	/* GPU timer */
	hipEvent_t start, stop;
	checkCUDA( hipEventCreate( &start ) );
    checkCUDA( hipEventCreate( &stop ) );
	checkCUDA( hipEventRecord( start, 0 ) );

	/* launch the kernel on the GPU */
	gpu_naive<<< blocks, threads >>>( size, hx, ht, sigma, idx, idy, k_beg, k_end, d_E, d_Hx, d_Hy );
	checkKERNEL();
	
	/* stop the timers */
	checkCUDA( hipEventRecord( stop, 0 ) );
	checkCUDA( hipEventSynchronize( stop ) );
	float gpuTime;
	checkCUDA( hipEventElapsedTime( &gpuTime, start, stop ) );

	printf("GPU naive calculation time %f ms\n", gpuTime );
	
	floatT *out_E, *out_Hx, *out_Hy;
	out_E = (floatT *) malloc (numbytes_E);
	out_Hx = (floatT *) malloc (numbytes_H);
	out_Hy = (floatT *) malloc (numbytes_H);

	checkCUDA( hipMemcpy( out_E, d_E, numbytes_E, hipMemcpyDeviceToHost ) );
	checkCUDA( hipMemcpy( out_Hx, d_Hx, numbytes_H, hipMemcpyDeviceToHost ) );
	checkCUDA( hipMemcpy( out_Hy, d_Hy, numbytes_H, hipMemcpyDeviceToHost ) );

	int success = 1;
	floatT diff, thresh=1e-6;
	for( int i = 0; i < size; i++ )	{
		for ( int j = 0; j<size; j++ ) {
			diff = abs(1.0-out_E[INDX(i,j,size)]/h_E[INDX(i,j,size)]);
			if ( diff>thresh ) {
				printf("error in E element %d, %d: CPU %e vs GPU %e\n",i,j,h_E[INDX(i,j,size)],out_E[INDX(i,j,size)] );
				success = 0;
				break;
			}
		}
	} 

	for( int i = 0; i < size; i++ )	{
		for ( int j = 0; j<size-1; j++ ) {
			diff = abs(1.0-out_Hx[INDX(i,j,size)]/h_Hx[INDX(i,j,size)]);
			if ( diff>thresh ) {
				printf("error in Hx element %d, %d: CPU %e vs GPU %e\n",i,j,h_Hx[INDX(i,j,size)],out_Hx[INDX(i,j,size)] );
				success = 0;
				break;
			} 
		}
	} 
	
	for( int i = 0; i < size-1; i++ )	{
		for ( int j = 0; j<size; j++ ) {
			diff = abs(1.0-out_Hy[INDX(i,j,size)]/h_Hy[INDX(i,j,size)]);
			if ( diff>thresh) {
				printf("error in Hy element %d, %d: CPU %e vs GPU %e\n",i,j,h_Hy[INDX(i,j,size)],out_Hy[INDX(i,j,size)] );
				success = 0;
				break;
			} 
		}
	} 

	
	if( success == 1 ) printf("PASS\n");
	else               printf("FAIL\n");

	free(h_E);
	free(h_Hx);
	free(h_Hy);	
	free(out_E);
	free(out_Hx);
	free(out_Hy);
	checkCUDA( hipFree( d_E ) );
	checkCUDA( hipFree( d_Hx ) );
	checkCUDA( hipFree( d_Hy ) );

	checkCUDA( hipDeviceSynchronize() );
	
	return 0;
}
