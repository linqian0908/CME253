
#include <hip/hip_runtime.h>
#include "stdio.h"
#include "math.h"
#include "stdlib.h"
#include "time.h"


/* macro to index a 1D memory array with 2D indices in column-major order */
#define INDX( row, col, ld ) ( ( (col) * (ld) ) + (row) )
#define FJ(n, hx, ht, sigma) 1000*(exp(-pow(((n-0.5)*ht/sigma-4),2))*sin(2*M_PI*(n-0.5)*hx/800/sqrt(2.0)))

// GPU macro
#define THREADS_PER_BLOCK 256
typedef float floatT;

__global__ void gpu_naive(const int size, const int x, const floatT t, const floatT sigma,
	const int idx, const int idy, const int k_beg, const int k_end, 
	floatT *e, floatT *hx, floatT *hy) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	floatT temp;
	
	for (int k=k_beg; k<=k_end; k++) {
		if (i>0 && i<(size-1) && j>0 && j<(size-1)) {
			temp = e[INDX(i,j,size)] + (hy[INDX(i,j,size-1)] - hy[INDX(i-1,j,size-1)])
				- (hx[INDX(i,j,size)] - hx[INDX(i,j-1,size)]);
			if (i==idx && j==idy) {
				temp -= FJ(k, x, t, sigma);
			}
			e[INDX(i,j,size)] = temp;
		}
		__syncthreads();

		if (i<(size-1) && j<size) {
			temp = hy[INDX(i,j,size-1)] + 0.5*(e[INDX(i+1,j,size)]-e[INDX(i,j,size)]);
			hy[INDX(i,j,size-1)] = temp;			
		}
		if (i<size && j<size-1) {
			temp = hx[INDX(i, j, size)] - 0.5 * (e[INDX(i, j+1, size)] - e[INDX(i, j, size)]);
			hx[INDX(i, j, size)] = temp;
		}
		__syncthreads();
	}
		
}

void host_fdtd(const int size, const int x, const floatT t, const floatT sigma,
    const int idx, const int idy, const int k_beg, const int k_end, 
    floatT *e, floatT *hx, floatT *hy) {
	for (int k = k_beg; k <= k_end; k++) {
		for (int i = 1; i < (size-1); i++) {
			for (int j = 1; j < (size-1); j++) {
				e[INDX(i, j, size)] += (hy[INDX(i, j, (size-1))] - hy[INDX(i-1, j, (size-1))])
				- (hx[INDX(i, j, size)] - hx[INDX(i, j-1, size)]);
			}
		}
		e[INDX(idx, idy, size)] -= FJ(k, x, t, sigma);


		for (int i = 0; i < (size-1); i++) {
			for (int j = 0; j < size; j++) {
				hy[INDX(i,j,(size-1))] += 0.5 * (e[INDX(i+1, j, size)] - e[INDX(i, j, size)]);
			}
		}

		for (int i = 0; i < size; i++) {
			for (int j = 0; j < (size-1); j++) {
				hx[INDX(i, j, size)] -= 0.5 * (e[INDX(i, j+1, size)] - e[INDX(i, j, size)]);
			}
		}
	}
}

int main(int argc, char *argv[]) {
	
	floatT L = 800.0;
	floatT hx = 1.0;
	floatT ht = hx/sqrt(2.0)/3;
 	floatT sigma = 200*ht;

	fprintf(stdout, "fj output is %f\n", FJ(500, hx, ht, sigma));

	int size = int(2*L/hx)+1;
	int idx = int(1.5*L/hx)+1;
	int idy = int(L/hx)+1;

	floatT *h_E, *h_Hx, *h_Hy;

	size_t num_E = size * size;
	size_t num_H = (size - 1)*size;	
	size_t numbytes_E = num_E*sizeof(floatT);
	size_t numbytes_H = num_H*sizeof(floatT);
	fprintf(stdout, "total memory allocated is %lu\n", numbytes_E+2*numbytes_H);
	
	clock_t t_begin, t_end;	
	t_begin = clock();
	h_E = (floatT *) calloc (num_E, sizeof(floatT));
	h_Hx = (floatT *) calloc (num_H, sizeof(floatT));
	h_Hy = (floatT *) calloc (num_H, sizeof(floatT));

	h_E[INDX(idx, idy, size)] = - FJ(1, hx, ht, sigma);
	t_end = clock();
	fprintf(stdout, "CPU memory allocation time is %f s\n", (float)(t_end - t_begin) / CLOCKS_PER_SEC);

	int k_beg = 2;
	int k_end = 1500;
	
	t_begin = clock();
	host_fdtd(size, hx, ht, sigma, idx, idy, k_beg, k_end, h_E, h_Hx, h_Hy);
	t_end = clock();
	fprintf(stdout, "CPU calculation time for %d iteration is %f s\n", k_end, (float)(t_end - t_begin) / CLOCKS_PER_SEC);
	
	// GPU memory allocation and initialization
	floatT *d_E, *d_Hx, *d_Hy;
	hipMalloc( (void **) &d_E, numbytes_E );
	hipMalloc( (void **) &d_Hx, numbytes_H );
	hipMalloc( (void **) &d_Hy, numbytes_H );

	hipMemcpy(d_E, h_E, numbytes_E, hipMemcpyHostToDevice);
	hipMemset(d_Hx, 0, numbytes_H);
	hipMemset(d_Hy, 0, numbytes_H);
	
	dim3 threads( THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
	dim3 blocks( (size/threads.x)+1, (size/threads.y)+1, 1);

	/* GPU timer */
	hipEvent_t start, stop;
	hipEventCreate( &start );
	hipEventCreate( &stop );
	hipEventRecord( start, 0 );

	/* launch the kernel on the GPU */
	gpu_naive<<< blocks, threads >>>( size, hx, ht, sigma, idx, idy, k_beg, k_end, d_E, d_Hx, d_Hy );

	/* stop the timers */
	hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );
	float gpuTime;
	hipEventElapsedTime( &gpuTime, start, stop );

	printf("GPU naive calculation time %f ms\n", gpuTime );
	
	floatT *out_E, *out_Hx, *out_Hy;
	out_E = (floatT *) malloc (numbytes_E);
	out_Hx = (floatT *) malloc (numbytes_H);
	out_Hy = (floatT *) malloc (numbytes_H);

	hipMemcpy( out_E, d_E, numbytes_E, hipMemcpyDeviceToHost );
	hipMemcpy( out_Hx, d_Hx, numbytes_H, hipMemcpyDeviceToHost );
	hipMemcpy( out_Hy, d_Hy, numbytes_H, hipMemcpyDeviceToHost );

	int success = 1;
	floatT diff, thresh=1e-9;
	for( int i = 0; i < size; i++ )	{
		for ( int j = 0; j<size; j++ ) {
			diff = abs(1.0-out_E[INDX(i,j,size)]/h_E[INDX(i,j,size)]);
			if ( diff>thresh ) {
				printf("error in E element %d, %d: CPU %f vs GPU %f\n",i,j,h_E[INDX(i,j,size)],out_E[INDX(i,j,size)] );
				success = 0;
				break;
			} /* end if */
		}
		if (success==0) { break;}
	} /* end for */
/*
	for( int i = 0; i < size; i++ )	{
		for ( int j = 0; j<size-1; j++ ) {
			diff = abs(1.0-out_Hx[INDX(i,j,size)]/h_Hx[INDX(i,j,size)]);
			if ( diff>thresh ) {
				printf("error in Hx element %d, %d: CPU %f vs GPU %f\n",i,j,h_E[INDX(i,j,size)],out_E[INDX(i,j,size)] );
				success = 0;
				break;
			} 
		}
		if (success==0) { break;}
	} 
	
	for( int i = 0; i < size-1; i++ )	{
		for ( int j = 0; j<size; j++ ) {
			diff = abs(1.0-out_Hy[INDX(i,j,size)]/h_Hy[INDX(i,j,size)]);
			if ( diff>thresh) {
				printf("error in Hy element %d, %d: CPU %f vs GPU %f\n",i,j,h_E[INDX(i,j,size)],out_E[INDX(i,j,size)] );
				success = 0;
				break;
			} 
		}
		if (success==0) { break;}
	} 
*/
	
	if( success == 1 ) printf("PASS\n");
	else               printf("FAIL\n");

	free(h_E);
	free(h_Hx);
	free(h_Hy);	
	free(out_E);
	free(out_Hx);
	free(out_Hy);
	hipFree( d_E );
	hipFree( d_Hx );
	hipFree( d_Hy );

	hipDeviceSynchronize();
	
	return 0;
}
